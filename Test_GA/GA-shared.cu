#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <algorithm>
#include <functional>
#include <iomanip>
#include <fstream>  
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include <sstream>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#include "constants.h"
using namespace std;
// assume block size equal population size
//ģ�庯������string���ͱ���ת��Ϊ���õ���ֵ���� 
template <class Type>
Type stringToNum(const string& str){
	istringstream iss(str);
	Type num;
	iss >> num;
	return num;
}

template <class T>
int getArrayLen(T& array)
{
	return (sizeof(array) / sizeof(array[0]) - 1);
}

const int THREADS_PER_BLOCK = 256*2;

void cudasafe(hipError_t error, char* message = "Error occured") {
	if(error != hipSuccess) {
		fprintf(stderr,"ERROR: %s : %i\n", message, error);
		exit(-1);
	}
}

__global__ void randomInit(hiprandState* state, unsigned long seed) {
    int tid = threadIdx.x;
    hiprand_init(seed, tid, 0, state + tid);
}

__device__ float fitness(M_args deviceParameter, M_args_Tset deviceParameter_Tset,float tau) {
    float result = 0;
	//printf("%d_a",deviceParameter.spike_data_num);
	//printf("%d_b", deviceParameter_Tset.length);

	for (size_t i = 0; i<deviceParameter.spike_data_num; ++i)
		for (size_t j = 0; j<deviceParameter_Tset.length; ++j)
	{
			//printf("%f_c ", deviceParameter.spike_data[i]);
			//printf("%f_d ", deviceParameter_Tset.spike_TestData[j]);
			result += expf(-fabsf(deviceParameter.spike_data[i] - deviceParameter_Tset.spike_TestData[j])*1.0/tau);
			//printf("%f_1 ", result);
       // ++curPos;
    }
	for (size_t i = 0; i<deviceParameter.spike_data_num; ++i)
		for (size_t j = 0; j<deviceParameter.spike_data_num; ++j)
		{
			result += expf(-fabsf(deviceParameter.spike_data[i] - deviceParameter.spike_data[j])*1.0 / tau);
			//printf("%f_2 ", result);
			// ++curPos;
		}
	for (size_t i = 0; i<deviceParameter_Tset.length; ++i)
		for (size_t j = 0; j<deviceParameter_Tset.length; ++j)
		{
			result -= 2*expf(-fabsf(deviceParameter_Tset.spike_TestData[i] - deviceParameter_Tset.spike_TestData[j])*1.0 / tau);
			//printf("%f_3 ", result);
			// ++curPos;
		}
	//printf("%f_4 ", result);
    return result;
}

__device__ float rastrigin(const float *curPos) {
    float result = 10.0f * VAR_NUMBER;
    for (size_t i=0; i<VAR_NUMBER; ++i) {
        result += *curPos * *curPos - 10.0f * cosf(2 * HIP_PI_F * *curPos);
        ++curPos;
    }
    return result;
}

////__global__ void GAKernel(float* population, float** sharedPopulation, float* sharedScore, M_args deviceParameter, M_args_Tset *deviceParameter_Tset, float tau) {
////	//__shared__ float sharedPopulation[THREADS_PER_BLOCK * 2][VAR_NUMBER];
////	//__shared__ float sharedScore[THREADS_PER_BLOCK * 2];
////
////
////	const int gid = blockDim.x * blockIdx.x + threadIdx.x;
////	const int tid = threadIdx.x;
////
////	M_args_Tset curPos = deviceParameter_Tset[tid];
////	sharedScore[tid] = fitness(deviceParameter, curPos, tau);
////	// loading initial random population into shared memory
////	if (gid < POPULATION_SIZE) {
////		for (int i = 0; i < VAR_NUMBER; ++i)
////			sharedPopulation[tid][i] = population[gid * VAR_NUMBER + i];
////	}
////
////	sharedScore[tid + THREADS_PER_BLOCK] = 123123.0;
////
////	__syncthreads();
////	//return 
////}
__global__ void GAKernel_GenEach(float* population, ScoreWithId* score, hiprandState* randomStates, M_args deviceParameter, M_args_Tset *deviceParameter_Tset, float tau, int genindex, int MaxGeneration) {
	// we first have to calculate the score for the first half of threads
	//const float *curPos = sharedPopulation[tid];
	__shared__ float sharedPopulation[THREADS_PER_BLOCK * 2][VAR_NUMBER];
	__shared__ float sharedScore[THREADS_PER_BLOCK * 2];
	const float SIGN[2] = { -1.0f, 1.0f };
	const float MULT[2] = { 1.0f, 0.0f };

	const int gid = blockDim.x * blockIdx.x + threadIdx.x;
	const int tid = threadIdx.x;

	// loading initial random population into shared memory
	if (gid < POPULATION_SIZE) {
		for (int i = 0; i<VAR_NUMBER; ++i)
			sharedPopulation[tid][i] = population[gid * VAR_NUMBER + i];
	}

	//sharedScore[tid + THREADS_PER_BLOCK] = 123123.0;

	__syncthreads();
	hiprandState &localState = randomStates[tid];
	M_args_Tset curPos = deviceParameter_Tset[tid];
	sharedScore[tid] = fitness(deviceParameter, curPos, tau);
	// calculating score for the second half of individuals
	M_args_Tset curPos_b = deviceParameter_Tset[tid + THREADS_PER_BLOCK];
	sharedScore[tid + THREADS_PER_BLOCK] = fitness(deviceParameter, curPos_b, tau);

	__syncthreads();

	// selection
	// first half of threads writes best individual into its position
	if (sharedScore[tid] > sharedScore[tid + THREADS_PER_BLOCK]) {
		for (int i = 0; i < VAR_NUMBER; ++i)
			sharedPopulation[tid][i] = sharedPopulation[tid + THREADS_PER_BLOCK][i];
		sharedScore[tid] = sharedScore[tid + THREADS_PER_BLOCK];
	}

	__syncthreads();

	// now we've got best individuals in the first half of sharedPopulation

	// crossovers
	const int first = hiprand_uniform(&localState) * THREADS_PER_BLOCK;
	const int second = hiprand_uniform(&localState) * THREADS_PER_BLOCK;

	const float weight = hiprand_uniform(&localState);
	for (int i = 0; i < VAR_NUMBER; ++i) {
		sharedPopulation[tid + THREADS_PER_BLOCK][i] = sharedPopulation[first][i] * weight + sharedPopulation[second][i] * (1.0f - weight);
	}

	__syncthreads();

	// mutations on second half of population
	if (hiprand_uniform(&localState) < 0.8) {
		const float order = (hiprand_uniform(&localState) * 17) - 15;
		for (int i = 0; i < VAR_NUMBER; ++i) {
			const float mult = MULT[hiprand_uniform(&localState) < 0.8f];
			const float sign = SIGN[hiprand_uniform(&localState) < 0.5f];
			const float order_deviation = (hiprand_uniform(&localState) - 0.5f) * 5;
			sharedPopulation[tid + THREADS_PER_BLOCK][i] += powf(10.0f, order + order_deviation) * sign * mult;
		}
	}

	//sharing a part of population with others
	if ((blockIdx.x + first) % 5 == 0) {
		for (int i = 0; i < VAR_NUMBER; ++i)
			population[gid * VAR_NUMBER + i] = sharedPopulation[tid][i];
	}

	// take some best individuals from neighbour
	if ((blockIdx.x + first) % 3 == 0) {
		if (hiprand_uniform(&localState) < 0.11) {
			const int anotherBlock = hiprand_uniform(&localState) * (POPULATION_SIZE / THREADS_PER_BLOCK);
			const int ngid = blockDim.x * anotherBlock + threadIdx.x;
			for (int i = 0; i < VAR_NUMBER; ++i)
				sharedPopulation[tid][i] = population[ngid * VAR_NUMBER + i];
			//sharedScore[tid] = fitness(sharedPopulation[tid], deviceParameter);
			//sharedScore[tid]=fitness(deviceParameter, curPos_b, tau);
		}
	}

	////// output current population back
	if (gid < POPULATION_SIZE) {
		for (int i = 0; i < VAR_NUMBER; ++i)
			population[gid * VAR_NUMBER + i] = sharedPopulation[tid][i];
		if (genindex>=MaxGeneration)
			score[gid].score = sharedScore[tid];
	}
}

void printPopulation(const float* devicePopulation, const ScoreWithId* deviceScore) {
	float population[POPULATION_SIZE][VAR_NUMBER];
	cudasafe(hipMemcpy(population, devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyDeviceToHost), "Could not copy population from device");

	ScoreWithId score[POPULATION_SIZE];
	cudasafe(hipMemcpy(score, deviceScore, POPULATION_SIZE * sizeof (ScoreWithId), hipMemcpyDeviceToHost), "Could not copy score to host");

	//std::cout.cetf(std::ios::fixed);
	std::cout.precision(12);
	
	for (int i=0; i<POPULATION_SIZE; ++i) {
		std::cout << std::setw(15) << i << ' ';
	}
	std::cout << std::endl;

	for (int i=0; i<VAR_NUMBER; i++) {
		for (int u=0; u<POPULATION_SIZE; ++u) {
			std::cout << std::setw(15) << population[u][i] << ' ';
		}
		std::cout << std::endl;
	}
	std::cout << "Score: " << std::endl;
	for (int i=0; i<POPULATION_SIZE; ++i) {
		std::cout << std::setw(15) << score[i].score << ' ';
	}
	std::cout << std::endl;
}
void printFinalPopulation(const float* devicePopulation, const ScoreWithId* deviceScore) {
	float population[POPULATION_SIZE][VAR_NUMBER];
	cudasafe(hipMemcpy(population, devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyDeviceToHost), "Could not copy population from device");

	ScoreWithId score[POPULATION_SIZE];
	cudasafe(hipMemcpy(score, deviceScore, POPULATION_SIZE * sizeof (ScoreWithId), hipMemcpyDeviceToHost), "Could not copy score to host");

	//std::cout.cetf(std::ios::fixed);
	std::cout.precision(12);

	for (int i = 0; i<1; ++i) {
		std::cout << std::setw(15) << i << ' ';
	}
	std::cout << std::endl;

	for (int i = 0; i<VAR_NUMBER; i++) {
		for (int u = 0; u<1; ++u) {
			std::cout << std::setw(15) << population[u][i] << ' ';
		}
		std::cout << std::endl;
	}
	std::cout << "Score: " << std::endl;
	for (int i = 0; i<1; ++i) {
		std::cout << std::setw(15) << score[i].score << ' ';
	}
	std::cout << std::endl;
}
double solveGPU(M_args Parameter_) {
    cudasafe(hipSetDevice(0), "Could not set device 0");

	double ans = 0;
	int MaxGeneration = 1;
	float tau = 12;
	//M_args *IndexParameter_ = new M_args[MaxGeneration];
	//IndexParameter_ = 0;
	//IndexParameter_[0] = Parameter_;
	///////////////////////////////



	//////////////////////////////////
	float *population = new float[POPULATION_SIZE * VAR_NUMBER];

	for (int i=0; i<POPULATION_SIZE; ++i) {
		for (int j=0; j<VAR_NUMBER; ++j) {
			population[i * VAR_NUMBER + j] = (float_random() - 0.5f) * 10;
		}
	}
	M_args_Tset *Parameter_Tset=new M_args_Tset[POPULATION_SIZE];

	// copying population to device
	float *devicePopulation = 0;
	float *nextGeneration = 0;
	M_args_Tset *deviceParameter_Tset = 0;
	ScoreWithId *deviceScore = 0;
	hiprandState* randomStates;
	M_args deviceParameter_;
	deviceParameter_.current_data_num = Parameter_.current_data_num;
	deviceParameter_.spike_data_num = Parameter_.spike_data_num;
	//int DataLength = getArrayLen(Parameter_.spike_data);


	//int DataLengthC = getArrayLen(Parameter_.current_data);
	//Parameter_.length = DataLength;

	cudasafe(hipMalloc(&randomStates, THREADS_PER_BLOCK * sizeof(hiprandState)), "Could not allocate memory for randomStates");
	cudasafe(hipMalloc((void **)&devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float)), "Could not allocate memory for devicePopulation");
	cudasafe(hipMalloc((void **)&nextGeneration, POPULATION_SIZE * VAR_NUMBER * sizeof(float)), "Could not allocate memory for nextGeneration");
	cudasafe(hipMalloc((void **)&deviceScore, POPULATION_SIZE * sizeof (ScoreWithId)), "Could not allocate memory for deviceScore");
	cudasafe(hipMalloc((void **)&deviceParameter_Tset, 2*POPULATION_SIZE * sizeof (M_args_Tset)), "Could not allocate memory for deviceParameter_Tset");
	cudasafe(hipMalloc((void **)&deviceParameter_.current_data, Parameter_.current_data_num*sizeof(float)), "Could not allocate memory for deviceParameter_");
	cudasafe(hipMalloc((void **)&deviceParameter_.spike_data, Parameter_.spike_data_num*sizeof(float)), "Could not allocate memory for deviceParameter_");
	//cudasafe(hipMalloc((void **)&deviceParameter_, sizeof(M_args)), "Could not allocate memory for deviceParameter_");
	//cudasafe(hipMalloc((void **)&deviceParameter_.spike_TestData, DataLength*sizeof(float)), "Could not allocate memory for deviceParameter_");

	cudasafe(hipMemcpy(devicePopulation, population, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyHostToDevice), "Could not copy population to device");
	cudasafe(hipMemcpy(deviceParameter_.current_data, Parameter_.current_data, Parameter_.current_data_num*sizeof(float), hipMemcpyHostToDevice), "Could not copy Parameter_current_data to device");
	cudasafe(hipMemcpy(deviceParameter_.spike_data, Parameter_.spike_data, Parameter_.spike_data_num*sizeof(float), hipMemcpyHostToDevice), "Could not copy Parameter_spike_data to device");

	//cudasafe(hipMemcpy(deviceParameter_.spike_TestData, Parameter_.spike_TestData, DataLength*sizeof(float), hipMemcpyHostToDevice), "Could not copy Parameter_ to device");

	// invoking random init
	randomInit<<<1, THREADS_PER_BLOCK>>>(randomStates, 900);
	cudasafe(hipGetLastError(), "Could not invoke kernel randomInit");
	cudasafe(hipDeviceSynchronize(), "Failed to syncrhonize device after calling randomInit");

	const int BLOCKS_NUMBER = (POPULATION_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;


	//__shared__ float sharedPopulation[THREADS_PER_BLOCK * 2][VAR_NUMBER];
	//__shared__ float sharedScore[THREADS_PER_BLOCK * 2];
    //for (int i=0; i<1115; i++) {
	//void GAKernel_GenEach(float* population, ScoreWithId* score, hiprandState* randomStates, M_args deviceParameter, M_args_Tset *deviceParameter_Tset, float tau)
	for (int k = 0; k < MaxGeneration; k++) 
	{
		for (int j = 0; j < POPULATION_SIZE; ++j)
		{
			float *temp_spike_TestData;
			
			Parameter_Tset[j].spike_TestData = HH_return(&population[j], VAR_NUMBER, Parameter_Tset[j].length);
			hipMalloc(&temp_spike_TestData, Parameter_Tset[j].length*sizeof(float));
			std::cout << Parameter_Tset[j].length << std::endl;
			cudasafe(hipMemcpy(&deviceParameter_Tset[j], &Parameter_Tset[j], sizeof (M_args_Tset), hipMemcpyHostToDevice), "Could not copy deviceParameter_Tset1 to device");
			cudasafe(hipMemcpy(temp_spike_TestData, Parameter_Tset[j].spike_TestData, (Parameter_Tset[j].length*sizeof(float)), hipMemcpyHostToDevice), "Could not copy deviceParameter_Tset_spike_TestData2 to device");

			cudasafe(hipMemcpy(&deviceParameter_Tset[j].spike_TestData, &temp_spike_TestData, sizeof(float*), hipMemcpyHostToDevice), "Could not copy deviceParameter_Tset_spike_TestData to device");
			cudasafe(hipFree(temp_spike_TestData), "Could not free temp_spike_TestData");
	
			
		}
		GAKernel_GenEach << <BLOCKS_NUMBER, THREADS_PER_BLOCK >> >(devicePopulation, deviceScore, randomStates, deviceParameter_, deviceParameter_Tset, tau, k, MaxGeneration);
		//delete[]population;
		//float *population = new float[POPULATION_SIZE * VAR_NUMBER];
		cudasafe(hipMemcpy(population, devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyDeviceToHost), "Could not copy population from device");

		printf("%d_1111\n", k);
		printFinalPopulation(devicePopulation, deviceScore);
		printf("%d_2222\n", k);
		////GAKernel_gen << <BLOCKS_NUMBER, THREADS_PER_BLOCK >> >(devicePopulation, sharedPopulation, sharedScore, deviceScore, randomStates, deviceParameter_, deviceParameter_Tset, tau);
	}
	cudasafe(hipGetLastError(), "Could not invoke GAKernel");
    cudasafe(hipDeviceSynchronize(), "Failed to syncrhonize device after calling GAKernel");

    //printPopulation(devicePopulation, deviceScore);
    //}

	// freeing memory
	cudasafe(hipFree(devicePopulation), "Failed to free devicePopulation");
	cudasafe(hipFree(deviceScore), "Failed to free deviceScore");
	cudasafe(hipFree(randomStates), "Could not free randomStates");
	cudasafe(hipFree(nextGeneration), "Could not free nextGeneration");
	cudasafe(hipFree(deviceParameter_Tset), "Could not free deviceParameter_Tset");
	delete[] population;

	return ans;
}

float * Read_Txt(string filename,int &num)
{
	float *Mdata = new float[100000];
	ifstream in(filename);
	string line;
	int i = 0;
	if (in) // �и��ļ�  
	{
		while (getline(in, line)) // line�в�����ÿ�еĻ��з�  
		{
			//cout << stringToNum<float>(line)+0.015 << endl;
			Mdata[i] = stringToNum<float>(line);
			i++;
		}
	}
	else // û�и��ļ�  
	{
		cout << "no such file" << endl;
		return 0;
	}
	num = i;
	float *Mdata_copy = new float[i];
	for (int j = 0; j < i; j++)
	{
		Mdata_copy[j] = Mdata[j];
	}
	return Mdata_copy;
}

int main() {
	freopen("output.txt", "w", stdout);
	srand(1900);
	srand(static_cast<unsigned>(time(0)));
	//float *spike_data, *current_data, *spike_TestData;
	M_args Parameter_;
	Parameter_.spike_data = Read_Txt("spikes.txt", Parameter_.spike_data_num);
	Parameter_.current_data = Read_Txt("current.txt", Parameter_.current_data_num);
	//Parameter_.spike_TestData = Read_Txt("spikes_test.txt");
	double ans = solveGPU(Parameter_);
	std::cout << "GPU answer = " << ans << std::endl;

	return 0;
}
